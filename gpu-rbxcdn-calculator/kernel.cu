﻿
#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

__device__ __host__ int compute_rbxcdn(int i, const char* hash);
__global__ void rbxcdn_kernel(const char* hash, int* output);

int main() {
    std::string hash_input;
    while (true) {
        std::cout << "Enter hash: ";
        std::getline(std::cin, hash_input);
        int* output;
        char* hash;
        hipMallocManaged(&hash, 33);
        hipMallocManaged(&output, sizeof(int));
        strncpy(hash, hash_input.c_str(), 32);
        hash[32] = '\0';

        rbxcdn_kernel <<<1, 1 >>> (hash, output);
        hipDeviceSynchronize();

        std::cout << "The designated rbxcdn bucket ID is: " << *output << std::endl;

        hipFree(hash);
        hipFree(output);
    }
}

__device__ __host__ int compute_rbxcdn(int i, const char* hash) {
    for (int t = 0; t < 32; t++)
        i = i ^ hash[t];
    return i % 8;
}

__global__ void rbxcdn_kernel(const char* hash, int* output) {
    *output = compute_rbxcdn(31, hash);
}